/*This file is part of waveeq project.

  waveeq project is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  waveeq project is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with waveeq project.  If not, see <http://www.gnu.org/licenses/>.
  */



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 10
#endif

int compute_wave_cpu(double *unew,double *ucur,double *uold,int nx,int ny,double dx,double dy,double dt,int niter,double cel)
{
	for(int iter = 0; iter < niter; iter++)
    {
		for (int i = 1; i < nx-1; i++)
			for (int j = 1; j < ny-1; j++)
            {
				unew[i*ny+j] =2.0*ucur[i*ny+j]-uold[i*ny+j]
						+cel*cel*dt*dt*((ucur[(i-1)*ny+j] - 2.0*ucur[i*ny+j] + ucur[(i+1)*ny+j])/(dx*dx)
                                        +(ucur[i*ny+(j-1)] - 2.0*ucur[i*ny+j] + ucur[i*ny+(j+1)])/(dy*dy));
			}

		/* Reflecting Conditions */
		for (int j = 0; j < ny; j++) unew[(nx-1)*ny+j] = ucur[(nx-3)*ny+j];
		for (int j = 0; j < ny; j++) unew[j] = ucur[2*ny+j];
		for (int i = 0; i < nx; i++) unew[i*ny] = ucur[i*ny+2];
		for (int i = 0; i < nx; i++) unew[i*ny+ny-1] = ucur[i*ny+ny-3];

		memcpy(uold, ucur, nx*ny*sizeof(double));
		memcpy(ucur, unew, nx*ny*sizeof(double));
	}

	return EXIT_SUCCESS;
}

__global__ void compute_wave_gpu(double * __restrict__ unew, const double * __restrict__ ucur,
                                 const double * __restrict__ uold, int nx, int ny,
                                 double dx, double dy, double dt,double cel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int inner = (0 < x && x < nx - 1 && 0 < y && y < ny - 1);

    if (inner)
        unew[x * ny + y] = 2.0 * ucur[x*ny+y] - uold[x*ny+y]
                           + cel*cel*dt*dt *
                             ((ucur[(x-1)*ny+y] - 2.0*ucur[x*ny+y] + ucur[(x+1)*ny+y]) / (dx*dx)
                              + (ucur[x*ny+(y-1)] - 2.0*ucur[x*ny+y] + ucur[x*ny+(y+1)]) / (dy*dy));

    /* Reflecting Conditions */
    if (x == (nx - 1)) unew[(nx - 1) * ny + y] = ucur[(nx - 3) * ny + y];
    if (x == 0) unew[y] = ucur[2 * ny + y];
    if (y == 0) unew[x * ny] = ucur[x * ny + 2];
    if (y == (ny - 1)) unew[x * ny + ny - 1] = ucur[x * ny + ny - 3];
}

__global__ void compute_wave_gpu2(double * __restrict__ unew, const double * __restrict__ ucur,
                                 const double * __restrict__ uold, int nx, int ny,
                                 double dx, double dy, double dt,double cel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // copy block in sm
    __shared__ double s_ucur[BLOCK_SIZE + 2][BLOCK_SIZE + 2];
    s_ucur[threadIdx.x + 1][threadIdx.y + 1] = ucur[x * ny + y];

    // copy halo in sm too
    if (threadIdx.x == 0 && 0 < x)                   s_ucur[0][threadIdx.y + 1] = ucur[(x-1) * ny + y];
    if (threadIdx.x == blockDim.x - 1 && x < nx-1)   s_ucur[BLOCK_SIZE + 1][threadIdx.y + 1] = ucur[(x+1) * ny + y];
    if (threadIdx.y == 0 && 0 < y)                   s_ucur[threadIdx.x + 1][0] = ucur[x * ny + (y-1)];
    if (threadIdx.y == blockDim.y - 1 && y < ny-1)   s_ucur[threadIdx.x + 1][BLOCK_SIZE + 1] = ucur[x * ny + (y+1)];

    // Synchronize to make sure the sub-matrices are loaded
    // before starting the computation
    __syncthreads();

    if (0 < x && x < nx-1 && 0 < y && y < ny-1)
        unew[x*ny+y] = 2.0 * s_ucur[1 + threadIdx.x][1 + threadIdx.y] - uold[x*ny+y]
                       +cel*cel*dt*dt*((s_ucur[threadIdx.x][1 + threadIdx.y] - 2.0*s_ucur[1 + threadIdx.x][1 + threadIdx.y] + s_ucur[2 + threadIdx.x][1 + threadIdx.y])/(dx*dx)
                                       +(s_ucur[1 + threadIdx.x][threadIdx.y] - 2.0*s_ucur[1 + threadIdx.x][1 + threadIdx.y] + s_ucur[1 + threadIdx.x][2 + threadIdx.y])/(dy*dy));

    /* Reflecting Conditions */
    if (x == 0) unew[y] = s_ucur[3][1 + threadIdx.y];
    if (y == 0) unew[x*ny] = s_ucur[1 + threadIdx.x][3];
    if (x == (nx-1)) unew[(nx-1)*ny+y] = s_ucur[BLOCK_SIZE - 1][1 + threadIdx.y];
    if (y == (ny-1)) unew[x*ny+(ny-1)] = s_ucur[1 + threadIdx.x][BLOCK_SIZE - 1];
}