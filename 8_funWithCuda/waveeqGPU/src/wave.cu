/*This file is part of waveeq project.

  waveeq project is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  waveeq project is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with waveeq project.  If not, see <http://www.gnu.org/licenses/>.
  */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#define _max(a,b) (a>b)?a:b;

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 10
#endif

void usage(char * argv[]);
int vtkprint(int index, int nx, int ny, double dx, double dy, double *uold);
int compute_wave_cpu(double *unew, double *ucur, double *uold, int nx, int ny,
                     double dx, double dy, double dt, int niter, double cel);

__global__ void compute_wave_gpu(double * __restrict__ unew, const double * __restrict__ ucur,
                                 const double * __restrict__ uold, int nx, int ny,
                                 double dx, double dy, double dt, double cel);

__global__ void compute_wave_gpu2(double * __restrict__ unew, const double * __restrict__ ucur,
                                  const double * __restrict__ uold, int nx, int ny,
                                  double dx, double dy, double dt, double cel);

int main(int argc,char* argv[])
{
    int sizex, sizey, nx, ny, niter;
    double x0,y0;
    double dt, dx, dy;
    const double sigma = 0.2;
    double *ucur;
    double *dunew, *ducur, *duold;
    double *tmp;


    const double cel = 4.0;

    struct timeval t1;
    struct timeval t2;
    double time;

    if(argc < 7) usage(argv);

    /* Struct definition */

    sizex = atoi(argv[1]);
    sizey = atoi(argv[2]);
    nx = atoi(argv[3]);
    ny = atoi(argv[4]);
    dt = (double) atof(argv[5]);
    niter = atoi(argv[6]);

    dx = (double) sizex / (double) nx;
    dy = (double) sizey / (double) ny;

    fprintf(stdout,"dx = %g and dy = %g \n",dx,dy);
    fprintf(stdout,"dt = %g \n",dt);

    /* Memory allocation */

    ucur = (double *)calloc(nx*ny,sizeof(double));
    hipMalloc(&dunew, nx*ny*sizeof(double));
    hipMalloc(&ducur, nx*ny*sizeof(double));
    hipMalloc(&duold, nx*ny*sizeof(double));

    /* Initialization */

    x0 = (double) ( nx ) / 2.0 ;
    y0 = (double) ( ny ) / 4.0 ;

    printf("x0 = %g -- y0 = %g \n",x0,y0);
    printf("nx = %i -- ny = %i \n",nx,ny);
    printf("CFL = cel * dt / dx = %g\n",cel*dt/dx);

    for (int i = 0; i < nx; i++)
    {
        for (int j = 0; j < ny; j++)
        {
            ucur[i*ny+j] = _max(0.0,1.0-1.0/(sigma*sqrt(6.3))*exp(-0.5*((double)((i-x0)*(i-x0)*dx*dx)/(sigma*sigma)+(double)((j-y0)*(j-y0)*dy*dy)/(sigma*sigma))) );
        }
    }
    vtkprint(0,nx,ny,dx,dy,ucur);

    hipMemcpy(ducur, ucur, nx*ny*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(duold, ducur, nx*ny*sizeof(double), hipMemcpyDeviceToDevice);

    /* Wave propagation */
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(nx/BLOCK_SIZE, ny/BLOCK_SIZE);
    gettimeofday(&t1, NULL);
    for (int iter = 0; iter < niter; iter += 100)
    {
        for (int i = 0; i < 100; i ++) {
            compute_wave_gpu2 <<< dimGrid, dimBlock >>> (dunew, ducur, duold, nx, ny, dx, dy, dt, cel);
            tmp = duold;
            duold = ducur;
            ducur = dunew;
            dunew = tmp;
        }

        hipMemcpy(ucur, ducur, nx*ny*sizeof(double), hipMemcpyDeviceToHost);
        vtkprint(iter+1,nx,ny,dx,dy,ucur);
    }
    gettimeofday(&t2, NULL);
    time =( (t2.tv_sec-t1.tv_sec) + (t2.tv_usec-t1.tv_usec)*1.0e-06 );
    printf("time to compute stencil   = %g s \n", time);

    /* Memory free */

    hipFree(dunew);
    hipFree(ducur);
    hipFree(duold);
    free(ucur);
    return EXIT_SUCCESS;

}


