
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void VecAdd(float *A, float *B, float *C)
{
	int i = threadIdx.x; 
	C[i] = A[i] + B[i];
}

void printVec(int N, float *vec)
{
	for (int i = 0; i < N; i++)
	{
		printf("%.2f ", vec[i]);
	}
	printf("\n");
}

int main(int argc, char const *argv[])
{
	int deviceCount, device; 
	hipGetDeviceCount(&deviceCount); 

	for (device = 0; device < deviceCount; ++device) 
	{ 
		hipDeviceProp_t deviceProp; 
		hipGetDeviceProperties(&deviceProp, device); 
		printf("Device %d (%s) has %d multiprocessors, and warps of size %d\n", 
			device, deviceProp.name, deviceProp.multiProcessorCount, deviceProp.warpSize);
	}

	int N = 50000;
	size_t size = N * sizeof(float);

	float *hA = (float *)malloc(size);
	float *hB = (float *)malloc(size);
	float *hC = (float *)malloc(size);

	for (int i = 0; i < N; i++)
	{
		hA[i] = i;
		hB[i] = -2*i;
	}

	printVec(10, hA);
	printf("+\n");
	printVec(10, hB);

	float *dA, *dB, *dC;
	hipMalloc(&dA, size);
	hipMalloc(&dB, size);
	hipMalloc(&dC, size);

	hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);

	VecAdd<<<4, N/4>>>(dA, dB, dC);

	hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

	printf("=\n");
	printVec(10, hC);

	hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    free(hA);
    free(hB);
    free(hC);
}