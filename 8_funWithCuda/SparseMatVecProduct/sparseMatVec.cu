#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

//#define DISPLAY

typedef struct MatCSR
{
	int M;
	int N;
	int *ptrs; // row ptrs + nbNZ, size M+1
	int *idx; // col indices, size nbNZ
	float *vals; // values, size nbNZ
} MatCSR;

static void
initRandomMatCSR(MatCSR *mat, int M, int N, float density, float min, float max)
{
	int nbNZ = M*N*density;
	mat->M = M;
	mat->N = N;
	mat->ptrs = (int *) malloc((M+1) * sizeof(int));
	mat->idx = (int *) malloc(nbNZ * sizeof(int));
	mat->vals = (float *) malloc(nbNZ * sizeof(float));

	int currentPtr = 0;
	int currentIdx = 0;
	for (int row = 0; row < M; row++)
	{
		mat->ptrs[currentPtr++] = currentIdx;
		for (int col = 0; col < N; col++)
		{
			if ((currentIdx < nbNZ) && ((float)rand() / (float)RAND_MAX) < density)
			{
				mat->idx[currentIdx] = col;
				mat->vals[currentIdx++] = min + (max - min) * ((float)rand() / (float)RAND_MAX);
			}
		}
	}
	mat->ptrs[currentPtr] = currentIdx;
}

static void
freeMatCSR(MatCSR *mat)
{
	free(mat->ptrs);
	free(mat->idx);
	free(mat->vals);
}

static void 
sparseGemv(float *y, MatCSR *A, float *x)
{
	float dot;
	int rowStart, rowEnd;

	for (int row = 0; row < A->M; row++)
	{
		dot = 0;
		rowStart = A->ptrs[row];
		rowEnd = A->ptrs[row+1]; 
		for (int i = rowStart; i < rowEnd; i++)
			dot += A->vals[i] * x[A->idx[i]];
		y[row] = dot;
	}
}

__global__ void sparseGemvGPU(float *y, int *ptrs, int *idx, float *vals, float *x)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    float dot = 0;
    for (int i = ptrs[row]; i < ptrs[row+1]; i++)
        dot += vals[i] * x[idx[i]];
    y[row] = dot;
}

__global__ void sparseGemvGPU2(float *y, int *ptrs, int *idx, float *vals, float *x)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int nbValues = ptrs[row+1] - ptrs[row];
    __shared__ float values[nbValues];
    for (int i = 0; i < nbValues; i++)
        values[i] = vals[ptrs[row] + i];

    float dot = 0;
    for (int i = 0; i < nbValues; i++)
        dot += values[i] * x[idx[ptrs[row] + i]];
    y[row] = dot;
}

static void
sparseToDense(float *denseMat, MatCSR *sparseMat)
{
	memset(denseMat, 0, sparseMat->M * sparseMat->N * sizeof(float));

	int rowStart, rowEnd;
	for (int row = 0; row < sparseMat->M; row++)
	{
		rowStart = sparseMat->ptrs[row];
		rowEnd = sparseMat->ptrs[row+1]; 
		for (int i = rowStart; i < rowEnd; i++)
			denseMat[sparseMat->idx[i] * sparseMat->M + row] = sparseMat->vals[i];
	}
}

static void 
printMatCSR(char *name, MatCSR *mat)
{
	#ifdef DISPLAY
		printf("%s:\n", name);
		printf("Ptr : ");
		for (int i = 0; i < mat->M+1; i++)
			printf("%d ", mat->ptrs[i]);
		
		printf("\nIdx : ");
		for (int i = 0; i < mat->ptrs[mat->M]; i++)
			printf("%d ", mat->idx[i]);

		printf("\nVals : ");
		for (int i = 0; i < mat->ptrs[mat->M]; i++)
			printf("%.2f ", mat->vals[i]);

		printf("\n");
	#endif
}

static void
printDenseMat(const char *name, int M, int N, float *mat)
{
	#ifdef DISPLAY
		printf("%s:\n", name);
		for (int row = 0; row < M; row++)
		{
			for (int col = 0; col < N; col++)
			{
				printf("%.2f ", mat[col * M + row]);
			}
			printf("\n");
		}
	#endif
}

static float norm2(int N, float *a, float *b)
{
    float res = 0;
    for (int i = 0; i < N; i++)
    {
        res += fabsf(a[i] - b[i]);
    }
    return sqrtf(res);
}

void testDenseCublasGemv(int N, float *denseMat, float *x, float *y)
{
    float *dDenseMat;
    float *dX;
    float *dY;
    hipMalloc(&dDenseMat, N*N*sizeof(float));
    hipMalloc(&dX, N*sizeof(float));
    hipMalloc(&dY, N*sizeof(float));

    hipblasHandle_t  handle;
    hipblasCreate(&handle);
    hipblasSetMatrix(N, N, sizeof(float), denseMat, N, dDenseMat, N);
    hipblasSetVector(N, sizeof(float), x, 1, dX, 1);
//    hipblasSetVector(N, sizeof(float), y, 1, dY, 1);
    float al=1.0f;
    float bet =0.0f;
    hipblasSgemv(handle, HIPBLAS_OP_N, N, N, &al, dDenseMat, N, dX, 1, &bet, dY, 1);
    hipblasGetVector(N, sizeof(float), dY, 1, y, 1);

    hipFree(dDenseMat);
    hipFree(dX);
    hipFree(dY);
}

void testSparseGPU1(int N, MatCSR *sparseMat, float *x, float *y)
{
    float *dX, *dY, *dVals;
    int *dPtrs, *dIdx;
    hipMalloc(&dPtrs, (N+1)*sizeof(int));
    hipMalloc(&dIdx, sparseMat->ptrs[N]*sizeof(int));
    hipMalloc(&dVals, sparseMat->ptrs[N]*sizeof(float));
    hipMalloc(&dX, N*sizeof(float));
    hipMalloc(&dY, N*sizeof(float));

    hipMemcpy(dPtrs, sparseMat->ptrs, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dIdx, sparseMat->idx, sparseMat->ptrs[N]*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dVals, sparseMat->vals, sparseMat->ptrs[N]*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dX, x, N*sizeof(float), hipMemcpyHostToDevice);

    sparseGemvGPU<<<10, N/10>>>(dY, dPtrs, dIdx, dVals, dX);
    hipMemcpy(y, dY, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dPtrs);
    hipFree(dIdx);
    hipFree(dVals);
    hipFree(dX);
    hipFree(dY);
}

void testSparseGPU2(int N, MatCSR *sparseMat, float *x, float *y)
{
    float *dX, *dY, *dVals;
    int *dPtrs, *dIdx;
    hipMalloc(&dPtrs, (N+1)*sizeof(int));
    hipMalloc(&dIdx, sparseMat->ptrs[N]*sizeof(int));
    hipMalloc(&dVals, sparseMat->ptrs[N]*sizeof(float));
    hipMalloc(&dX, N*sizeof(float));
    hipMalloc(&dY, N*sizeof(float));

    hipMemcpy(dPtrs, sparseMat->ptrs, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dIdx, sparseMat->idx, sparseMat->ptrs[N]*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dVals, sparseMat->vals, sparseMat->ptrs[N]*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dX, x, N*sizeof(float), hipMemcpyHostToDevice);

    sparseGemvGPU2<<<10, N/10>>>(dY, dPtrs, dIdx, dVals, dX);
    hipMemcpy(y, dY, N*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dPtrs);
    hipFree(dIdx);
    hipFree(dVals);
    hipFree(dX);
    hipFree(dY);
}

int main(int argc, char const *argv[])
{
	srand(42);

	int N = 100;
	MatCSR sparseMat;
	float *denseMat = (float*) malloc(N * N * sizeof(float));
	float *x = (float *) malloc(N * sizeof(float));
    float *y = (float *) malloc(N * sizeof(float));
    float *yRef = (float *) malloc(N * sizeof(float));
	for (int i = 0; i < N; i++)
		x[i] = i+1;
	struct timeval start, stop;

	initRandomMatCSR(&sparseMat, N, N, 0.1, -20, 20);
	sparseToDense(denseMat, &sparseMat);
	printMatCSR("A sparse", &sparseMat);
	printDenseMat("A dense", N, N, denseMat);
	printDenseMat("x", 10, 1, x);

	printf("CPU:\n");
    gettimeofday(&start, NULL);
    sparseGemv(y, &sparseMat, x);
    gettimeofday(&stop, NULL);
	printDenseMat("y", N, 1, y);
	printf("duration = %e s\n", stop.tv_sec - start.tv_sec + 0.000001 * (stop.tv_usec - start.tv_usec));

    memcpy(yRef, y, N * sizeof(float));

	printf("GPU with CUblas :\n");
	gettimeofday(&start, NULL);
    testDenseCublasGemv(N, denseMat, x, y);
    gettimeofday(&stop, NULL);
	printDenseMat("y", N, 1, y);
	printf("duration = %e s\n", stop.tv_sec - start.tv_sec + 0.000001 * (stop.tv_usec - start.tv_usec));
    printf("error = %e s\n", norm2(N, y, yRef));

    printf("GPU custom sparse implementation 1:\n");
    gettimeofday(&start, NULL);
    testSparseGPU1(N, &sparseMat, x, y);
    gettimeofday(&stop, NULL);
    printDenseMat("y", N, 1, y);
    printf("duration = %e s\n", stop.tv_sec - start.tv_sec + 0.000001 * (stop.tv_usec - start.tv_usec));
    printf("error = %e s\n", norm2(N, y, yRef));

    printf("GPU custom sparse implementation 2:\n");
    gettimeofday(&start, NULL);
    testSparseGPU2(N, &sparseMat, x, y);
    gettimeofday(&stop, NULL);
    printDenseMat("y", N, 1, y);
    printf("duration = %e s\n", stop.tv_sec - start.tv_sec + 0.000001 * (stop.tv_usec - start.tv_usec));
    printf("error = %e s\n", norm2(N, y, yRef));


    freeMatCSR(&sparseMat);
	free(denseMat);
	free(x);
	free(y);
}